
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

#define N 1024  // Matrix size
#define M 1024  // Matrix size

int main() {
    double A[N*M], B[M*N], C[N*N];

    // Initialize matrices A and B (example values)
    // TODO: Modify this to match what Mahesh has for initialization.
    for (int i = 0; i < N*M; ++i) {
        A[i] = i;
        B[i] = i;
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * M * sizeof(double));
    hipMalloc(&d_B, M * N * sizeof(double));
    hipMalloc(&d_C, N * N * sizeof(double));

    hipblasSetMatrix(N, M, sizeof(double), A, N, d_A, N);
    hipblasSetMatrix(M, N, sizeof(double), B, M, d_B, M);

    const double alpha = 1.0;
    const double beta = 0.0;

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, M, &alpha, d_A, N, d_B, M, &beta, d_C, N);

    hipblasGetMatrix(N, N, sizeof(double), d_C, N, C, N);

/*    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }*/

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
